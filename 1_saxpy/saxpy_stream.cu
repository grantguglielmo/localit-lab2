#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "saxpy.h"

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < N; index += blockDim.x*gridDim.x){
       result[index] = alpha * x[index] + y[index];
    }
}

void
saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {

    const int threadsPerBlock = 512; // change this if necessary

    float *device_x;
    float *device_y;
    float *device_result;

    //
    // TODO: allocate device memory buffers on the GPU using
    // hipMalloc.  The started code issues warnings on build because
    // these buffers are used in the call to saxpy_kernel below
    // without being initialized.
    //
    hipMalloc(&device_x, total_elems*sizeof(float));
    hipMalloc(&device_y, total_elems*sizeof(float));
    hipMalloc(&device_result, total_elems*sizeof(float));
    long size = total_elems/partitions;
    hipStream_t streams[partitions];

    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

    for (int i=0; i<partitions; i++) {
  
        //
        // TODO: copy input arrays to the GPU using hipMemcpy
        //
        hipStreamCreate(&streams[i]);
        long index = size*i;
        hipMemcpy(device_x + index, xarray + index, size*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_y + index, yarray + index, size*sizeof(float), hipMemcpyHostToDevice);
         
        //
        // TODO: insert time here to begin timing only the kernel
        //
        double startGPUTime = CycleTimer::currentSeconds();
    
        // compute number of blocks and threads per block
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

        // run saxpy_kernel on the GPU
        saxpy_kernel<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(size, alpha, device_x + index, device_y + index, device_result + index);
    
        //
        // TODO: insert timer here to time only the kernel.  Since the
        // kernel will run asynchronously with the calling CPU thread, you
        // need to call hipDeviceSynchronize() before your timer to
        // ensure the kernel running on the GPU has completed.  (Otherwise
        // you will incorrectly observe that almost no time elapses!)
        //
        hipStreamSynchronize(streams[i]);
        double endGPUTime = CycleTimer::currentSeconds();
        double timeKernel = endGPUTime - startGPUTime;
    
        hipError_t errCode = hipPeekAtLastError();
        if (errCode != hipSuccess) {
            fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
        }
    
        //
        // TODO: copy result from GPU using hipMemcpy
        //
        hipMemcpy(resultarray + index, device_result + index, size*sizeof(float), hipMemcpyDeviceToHost);
    }

    // end timing after result has been copied back into host memory.
    // The time elapsed between startTime and endTime is the total
    // time to copy data to the GPU, run the kernel, and copy the
    // result back to the CPU
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    totalTimeAvg   += overallDuration;

    //
    // TODO free memory buffers on the GPU
    //
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
